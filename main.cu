#include "hip/hip_runtime.h"
/*
 * main.cpp
 *
 *  Created on: 29.4.2012
 *      Author: jirka
 */

#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <cmath>
#include <string.h>
#include <limits.h>
#include <map>
#include "Zasobnik.h"
#include "Graf.h"

using namespace std;

#define CHECK_AMOUNT  100

int POCET_VLAKEN = 32;

// const int NEJMENSI_MOZNY_STUPEN = 2;
int pocetUzlu;
int nejmensiStupen = INT_MAX;
int * zatimNejlepsiKostra = NULL;

int * zasobnik = NULL;
int ZASOBNIK_VELIKOST; // velikost zasobniku v poctu koster
int ZASOBNIK_POCATEK;
int ZASOBNIK_POCET_KOSTER;
int VELIKOST_KOSTRY; // velikost kostry v intech

bool jeZasobnikPrazdny() {
	return ZASOBNIK_POCET_KOSTER == 0;
}

void vytvorZasobnik(int velikost) {
	ZASOBNIK_VELIKOST = velikost;
	zasobnik = new int[velikost * VELIKOST_KOSTRY];
	ZASOBNIK_POCET_KOSTER = 0;
	ZASOBNIK_POCATEK = 0;
}

void roztahniZasobnik() {
	cout << "zvetsuji pole..." << endl;
	ZASOBNIK_VELIKOST *= 2 * VELIKOST_KOSTRY;
	int * novePole = new int [ZASOBNIK_VELIKOST];
	memcpy(novePole, zasobnik + ZASOBNIK_POCATEK,
			ZASOBNIK_POCET_KOSTER * VELIKOST_KOSTRY * sizeof(int));
	delete[] zasobnik;
	zasobnik = novePole;
	ZASOBNIK_POCATEK = 0;
}

void pridejKostruNaZasobnik(int * kostra, int pocatecniUzel, int koncovyUzel) {
	cout << "stack-add: " << (*kostra) << " + [" << pocatecniUzel << ","
			<< koncovyUzel << "]" << endl;
	if (ZASOBNIK_POCET_KOSTER == ZASOBNIK_VELIKOST) {
		roztahniZasobnik();
	}
	int index = (ZASOBNIK_POCATEK + ZASOBNIK_POCET_KOSTER) % ZASOBNIK_VELIKOST;
	index *= VELIKOST_KOSTRY;
	int velikostPoleKostry = 0;
	if (kostra != NULL) {
		while (kostra[velikostPoleKostry] != -1)
			velikostPoleKostry += 2;
	}
	memcpy(zasobnik + index, kostra, velikostPoleKostry * sizeof(int));
	zasobnik[index + velikostPoleKostry] = pocatecniUzel;
	zasobnik[index + velikostPoleKostry + 1] = koncovyUzel;
	if (velikostPoleKostry + 2 < VELIKOST_KOSTRY)
		zasobnik[index + velikostPoleKostry + 2] = -1;
	++ZASOBNIK_POCET_KOSTER;
}

 __device__ int pocetUzluVKostre(int * kostra, int velikost) {
	int pocet = 0;
	for (int i = 0; i < velikost && kostra[i] != -1; i += 2) {
		pocet++;
	}
	return ++pocet;
}

 __device__ int stupenKostry(int * kostra, int velikost) {
	int pole[100];
	for (int i = 0; i < 50; i++) {
		pole[i] = 0;
	}

	for (int i = 0; i < velikost && kostra[i] != -1; i++) {
		++(pole[kostra[i]]);
	}

	int nejStupen = 0;

	for (int i = 0; i < 50; i++) {
		if (pole[i] > nejStupen)
			nejStupen++;
	}
	return nejStupen;
}

 __device__ int stupenKostry(int * kostra, int pocatekDalsiHrany, int konecDalsiHrany,
		int velikost) {
	int pole[100];
		for (int i = 0; i < 50; i++) {
			pole[i] = 0;
		}

		for (int i = 0; i < velikost && kostra[i] != -1; i++) {
			++(pole[kostra[i]]);
		}
		++pole[pocatekDalsiHrany];
		++pole[konecDalsiHrany];


		int nejStupen = 0;

		for (int i = 0; i < 50; i++) {
			if (pole[i] > nejStupen)
				nejStupen++;
		}
		return nejStupen;
}

int * nactiRadek(string s, int nodes) {
	int * radek = new int[nodes];
	istringstream is(s);
	cout << s << ": ";

	for (int i = 0; i < nodes; i++) {
		radek[i] = s[i];
		cout << radek[i] << " ";
	}

	return radek;
}

int ** nactiGraf(char * soubor) {
	string s;
	ifstream in;
	in.open(soubor, ios::binary);
	getline(in, s);
	istringstream is(s);
	is >> pocetUzlu;

	//    in >> pocetUzlu;
	cout << "pocetUzlu: " << pocetUzlu << endl;
	//    in.ignore(INT_MAX, '\n');

	const int ** maticeSousednosti = new const int *[pocetUzlu];

	for (int i = 0; i < pocetUzlu; i++) {
		getline(in, s);
		int * radek = new int[pocetUzlu];
		//        cout << s << ": ";

		for (int j = 0; j < pocetUzlu; j++) {
			if (s.at(j) == '1')
				radek[j] = 1;
			else
				radek[j] = 0;
			cout << radek[j] << " ";
		}

		maticeSousednosti[i] = radek;
		cout << endl;
	}
	in.close();

	//return maticeSousednosti;


	int ** graf;
	    graf = new int * [pocetUzlu];
	    int i, j, k, sousedu;
	    for (i = 0; i < pocetUzlu; i++) {
	        sousedu = 0;
	        for (j = 0; j < pocetUzlu; j++) {
	            if (maticeSousednosti[i][j] == 1)
	                sousedu++;
	        }
	        graf[i] = new int[sousedu + 1];
	        sousedu = 0;
	        for (k = 0; k < pocetUzlu; k++) {
	            if (maticeSousednosti[i][k] == 1)
	                graf[i][sousedu++] = k;
	        }
	        graf[i][sousedu] = -1;
	    }

	    for (int m = 0; m < pocetUzlu; m++) {
	    	delete [] maticeSousednosti[m];
	    }

	    delete [] maticeSousednosti;
	    VELIKOST_KOSTRY = (pocetUzlu - 1) * 2;
	    return graf;
}

void vytiskniKostru() {
	cout << endl;
	cout << "KOSTRA S NEJNIZSIM STUPNEM:" << endl;
	cout << "===========================" << endl;
	cout << (*zatimNejlepsiKostra) << endl;
	cout << endl;
	cout << "NEJVYSSI STUPEN TETO KOSTRY:" << endl;
	cout << "============================" << endl;
	cout << nejmensiStupen << endl;
}

int pocetVychazejicichHran(int ** graf, int uzel) {
	int pocetHran = 0;
	for (int i = 0; i < pocetUzlu; i++) {
		if (graf[uzel][i] == -1)
			return pocetHran;
		else
			pocetHran++;
	}
	return pocetHran;
}

int * sousedniUzly(int ** graf, int uzel, int& pocet) {
	pocet = pocetVychazejicichHran(graf, uzel);
	return graf[uzel];
}

bool obsahujeUzel(int * kostra, int uzel, int velikostKostry) {
	for (int i = 0; i < velikostKostry; i++) {
		if (kostra[i] == -1) {
			return false;
		}
		if (kostra[i] == uzel) {
			return true;
		}
	}
	return false;
}

int zkusExpandovatZUzlu(int ** graf, int * kostra, int uzel,
		int velikostKostry) {
	int pocetSousedu;
	int pocet = 0;
	int * sousedi = sousedniUzly(graf, uzel, pocetSousedu);
	for (int i = 0; i < pocetSousedu; i++) {
		if (!obsahujeUzel(kostra, sousedi[i], velikostKostry)) {
			pocet++;
		}
	}
	return pocet;
}

int posledniPridanyUzelVKostre(int * kostra, int velikostKostry) {
	if (kostra[0] == -1)
		return -1;
	for (int i = 2; i < velikostKostry; i += 2) {
		if (kostra[i] == -1)
			return kostra[i - 1];
	}
	return kostra[velikostKostry];
}

 __device__ int predchoziUzel(int * kostra, int uzel, int velikostKostry) {
	int i = velikostKostry;
	for (int i = 0; i < velikostKostry; i += 2) {
		if (kostra[i] == -1) {
			i -= 1;
			break;
		}
	}
	for (; i > 0; i -= 2) {
		if (kostra[i] == uzel)
			return kostra[i - 1];
	}
	return -1;
}

 __device__ int expandujZUzlu(int ** graf, int * kostra, int uzel, int * zasobnik,
		int writeIndex, int velikostZasobniku, int velikostKostry) {
	int pocet;
	int * sousedi = sousedniUzly(graf, uzel, pocet);
	int i = 0;
	for (; i < pocet; i++) {
		if (!obsahujeUzel(kostra, sousedi[i], velikostKostry)) {
			if (stupenKostry(kostra, uzel, sousedi[i], velikostKostry)
					< nejmensiStupen) { // perspektivni reseni
				int j = 0;
				for (; j < velikostKostry; j++) {
					if (kostra[j] == -1) {
						break;
					}
					zasobnik[((writeIndex + i) % velikostZasobniku)
							* velikostKostry + j] = kostra[j];
				}
				zasobnik[((writeIndex + i) % velikostZasobniku) * velikostKostry
						+ j] = uzel;
				zasobnik[(writeIndex + i % velikostZasobniku) * velikostKostry
						+ j + 1] = sousedi[i];
				if (j + 2 < velikostKostry)
					zasobnik[((writeIndex + i) % velikostZasobniku)
							* velikostKostry + j + 2] = -1;
			}
		}
	}
	return ((writeIndex + i) % velikostZasobniku);
}

__device__ int zkusExpandovat(int ** graf, int * kostra, int velikostKostry) {
	int uzel = posledniPridanyUzelVKostre(kostra, velikostKostry);
	int pocet = zkusExpandovatZUzlu(graf, kostra, uzel, velikostKostry);
	if (pocet == 0) {
		uzel = predchoziUzel(kostra, uzel, velikostKostry);
		while (uzel > -1) {
			cout << "vracim se o hranu zpet..." << endl;
			pocet += zkusExpandovatZUzlu(graf, kostra, uzel, velikostKostry);
			uzel = predchoziUzel(kostra, uzel, velikostKostry);
		}
	}
	return pocet;
}

__device__ void expanduj(int ** graf, int * kostra, int velikostKostry, int * zasobnik, int velikostZasobniku, int writeIndex) {
	int uzel = posledniPridanyUzelVKostre(kostra, velikostKostry);
	int index = expandujZUzlu(graf, kostra, uzel, zasobnik, writeIndex, velikostZasobniku, velikostKostry);
	if (index == writeIndex) {
		uzel = predchoziUzel(kostra, uzel, velikostKostry);
		while (uzel > -1) {
			cout << "vracim se o hranu zpet..." << endl;
			index = expandujZUzlu(graf, kostra, uzel, zasobnik, index, velikostZasobniku, velikostKostry);
			uzel = predchoziUzel(kostra, uzel, velikostKostry);
		}
	}
}

__global__ void kernel(int ** graf, int** zasobnik, int * pocatekZasobniku,
		int * pocetKosterNaZasobniku, int * velikostZasobniku,
		int * reseni_kostry, int * hotovo, int velikostKostry, int checkInterval) {
	int citac = 0;
	bool konec = false;
	int myIndex = threadIdx.x;
	int index;
	int pocetNovychStavu = 0;
	int * kostra;
	__shared__	int reseni_stupne[32];

	__shared__	int cache[32];

	__shared__	int kostry[32];

	kostra = kostry + myIndex;

	reseni_stupne[myIndex] = INT_MAX;

	while (!konec) {
		citac++;
		index = (((*pocatekZasobniku) + myIndex) % (*velikostZasobniku)) * velikostKostry;
		if ((*zasobnik)[index] != -1) {
			// zkopirujeme si kostru do lok. pameti
			for (int i = 0; i < velikostKostry; i++) {
				kostra[i] = (*zasobnik)[index + i];
			}
			if (pocetUzluVKostre(kostra, velikostKostry)
					== pocetUzlu) {
				int stupen = stupenKostry(kostra, velikostKostry);
				if (stupen < reseni_stupne[myIndex]) {
					reseni_stupne[myIndex] = stupen;
					for (int i = 0; i < velikostKostry; i++) {
						reseni_kostry[myIndex * velikostKostry + i] =
								kostra[i];
						if (kostra[i] == -1)
							break;
					}
				}
				if (stupen == 2) {
					(*hotovo) = 1;
				}
				pocetNovychStavu = 0;
			} else {
				pocetNovychStavu = zkusExpandovat(graf,kostra,velikostKostry);
			}
			// odebereme kostru ze zasobniku
			(*zasobnik)[index] = -1;
		} else {
			if (myIndex == 0) {
				(*hotovo) = 1;
			}

		}

		if (citac % checkInterval == 0) {
			if ((*hotovo) == 1)
				konec = true;
		}

		cache[myIndex] = pocetNovychStavu;
		__syncthreads();
		int neaktivnich = 1;

		for (int i = 0; neaktivnich < blockDim.x; i++) {
			if (myIndex < blockDim.x - neaktivnich)
				cache[myIndex] += cache[myIndex + neaktivnich];
			__syncthreads();
			neaktivnich <<= 1;
		}

		if (myIndex == 0) {
			(*pocatekZasobniku) += blockDim.x;
			(*pocatekZasobniku) %= (*velikostZasobniku);
			(*pocetKosterNaZasobniku) = ((*pocetKosterNaZasobniku) < blockDim.x) ? 0 : (*pocetKosterNaZasobniku) - blockDim.x;



//			int staraVelikost = (*velikostZasobniku);
			if ((*pocetKosterNaZasobniku) + cache[0] > (*velikostZasobniku)) {
				cout << "pocet stavu: " << ((*pocetKosterNaZasobniku) + cache[0]) << endl;
			}
//			int * novyZas;
//			hipMalloc ((void**) &novyZas, (*velikostZasobniku) * velikostKostry * sizeof(int));
//		    hipMemcpy (novyZas, (*zasobnik), staraVelikost * VELIKOST_KOSTRY * sizeof(int), hipMemcpyDeviceToDevice);
//			hipFree(*zasobnik);
//		    (*zasobnik) = novyZas;
		}

		__syncthreads();

		int writeIndex = ((*pocetKosterNaZasobniku) + cache[myIndex])
				% ZASOBNIK_VELIKOST;

		expanduj(graf, kostra, velikostKostry, *zasobnik, *velikostZasobniku, writeIndex);

		__syncthreads();

		if (myIndex == 0) {
			(*pocetKosterNaZasobniku) += cache[0];
		}
		__syncthreads();
	}

	cache[myIndex] = reseni_stupne[myIndex];
	int step = blockDim.x / 2;

	while (step != 0) {
		if (myIndex < step)
			cache[myIndex] =
					cache[myIndex] < cache[myIndex + step] ?
							cache[myIndex] : cache[myIndex + step];
		__syncthreads();
		step /= 2;
	}

	if (myIndex == 0) {
		int nejStupen = cache[0];
		cout << "nejlepsi reseni: stupen " + nejStupen;
		for (int i = 0; i < blockDim.x; i++) {
			if (reseni_stupne[i] == nejStupen) {
				for (int j = 0; j < velikostKostry; j += 2) {
					cout << "[" << reseni_kostry[i * velikostKostry + j] << ", "
							<< reseni_kostry[i * velikostKostry + j] << "]";
					if (j < velikostKostry - 2)
						cout << " ";
				}
			}
		}
	}

}

int main(int argc, char *argv[]) {

	if (argc != 2) {
		cout << "Jako parametr zadejte nazev souboru." << endl;
		return (1);
	}

	int ** graf = nactiGraf(argv[1]);

	vytvorZasobnik(100000);

	int * reseni_kostry = new int[POCET_VLAKEN * VELIKOST_KOSTRY];

	// kopirovani grafu na GPU
	int ** g_graf;
	hipMalloc ((void**) &g_graf, pocetUzlu * sizeof(int*));
	for (int i = 0; i < pocetUzlu; i++) {
		int size = 0;
		for (int j = 0;; j++) {
			++size;
			if (graf[i][j] == -1)
				break;
		}
		hipMalloc ((void**) &g_graf[i], size * sizeof(int));
		hipMemcpy (g_graf, graf, size * sizeof(int), hipMemcpyHostToDevice);
	}

	int * sousedi = graf[0];
	if (sousedi[0] == -1) {
		cout << "Graf neni souvisly nebo neobsahuje alespon dva uzly" << endl;
		return (1);
	}
	for (int i = 0; sousedi[i] > -1; i++) {
		pridejKostruNaZasobnik(NULL, 0, sousedi[i]);
	}

	// kopirovani zasobniku na GPU
		int * g_zasobnik;
		hipMalloc ((void**) &g_zasobnik, ZASOBNIK_VELIKOST * VELIKOST_KOSTRY * sizeof(int));
		hipMemcpy (g_zasobnik, zasobnik, ZASOBNIK_VELIKOST * VELIKOST_KOSTRY * sizeof(int), hipMemcpyHostToDevice);

		int * g_reseni_kostry;
				hipMalloc ((void**) &g_reseni_kostry, POCET_VLAKEN * VELIKOST_KOSTRY * sizeof(int));


		int * g_pocatekZasobniku;
		int * g_pocetKoster;
		int * g_velikostZasobniku;
		int * hotovo;
		int * zasobnikPtr;

		hipMalloc ((void**) &g_pocatekZasobniku, sizeof(int));
		hipMalloc ((void**) &g_pocetKoster, sizeof(int));
		hipMalloc ((void**) &g_velikostZasobniku, sizeof(int));
		hipMalloc ((void**) &hotovo, sizeof(int));
		hipMalloc ((void**) &zasobnikPtr, sizeof(int*));
		(*g_pocatekZasobniku) = 0;
		(*g_pocetKoster) = ZASOBNIK_POCET_KOSTER;
		(*g_velikostZasobniku) = ZASOBNIK_VELIKOST;
		(*hotovo) = 0;
		zasobnikPtr = zasobnik;

		kernel<<<1, POCET_VLAKEN>>> (g_graf, &zasobnikPtr, g_pocatekZasobniku, g_pocetKoster, g_velikostZasobniku,
				g_reseni_kostry, hotovo, VELIKOST_KOSTRY, 100);

}

